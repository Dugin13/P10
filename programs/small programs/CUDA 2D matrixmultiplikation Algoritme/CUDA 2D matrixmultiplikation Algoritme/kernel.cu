
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void KernelFunc(const int **A, const int **B, int **C, const int Size)
{
}

int main()
{
	const int Size = 3;
	const int Size1d = Size*Size;
	int A[Size][Size];
	int B[Size][Size];
	int C[Size][Size];
	int i, x, y, z;

	for (int i = 0; i < Size; i++)
	{
		for (int x = 0; x < Size; x++)
		{
			A[i][x] = 2;
			B[i][x] = 3;
			C[i][x] = 0;
		}
	}
#pragma region normal version
	hipDeviceProp_t prop;
	int CountOfDevice;

	hipGetDeviceCount(&CountOfDevice);
	hipGetDeviceProperties(&prop, 0);
	if (prop.maxThreadsPerBlock > Size1d)
	{
		dim3 BlockSize(Size, Size);

	}
	else
	{

	}

#pragma endregion
	for (int i = 0; i < Size; i++)
	{
		for (int x = 0; x < Size; x++)
		{
			std::cout << "A: " << A[i][x] << "	B: " << B[i][x] << "	c: " << C[i][x] << std::endl;
		}
	}
	int STOP = 0;
	return 0;

    return 0;
}