
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

__global__ void KernelFunc(const int *A, const int *B, int *C, const int Size)
{
    int i = threadIdx.x;
	int x = i / Size;
	int y = i % Size;
	//D[i] = (x*Size) + y;
	for (int z = 0; z < Size; z++)
	{
		C[i] += A[(x*Size) + z] * B[(z*Size) + y];
	}
}



int main()
{
	const int Size = 3;
	const int Size1d = Size*Size;
	int A[Size1d];
	int B[Size1d];
	int C[Size1d];
	int i, x, y, z;

	for (int i = 0; i < Size1d; i++)
	{
		A[i] = 2;
		B[i] = 3;
		C[i] = 0;
		std::cout << A[i] << " " << B[i] << std::endl;
	}
#pragma region D1 version
	int *GPU_A, *GPU_B, *GPU_C;

	hipMalloc((void**)&GPU_A, Size1d * sizeof(int));
	hipMalloc((void**)&GPU_B, Size1d * sizeof(int));
	hipMalloc((void**)&GPU_C, Size1d * sizeof(int));

	hipMemcpy(GPU_A, A, Size1d * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPU_B, B, Size1d * sizeof(int), hipMemcpyHostToDevice);

	// TODO: lab en bedre m�de at g�re dette p�
	int threadsPerBlock = Size1d; 
	int blocksPerGrid = 1;

	KernelFunc <<<blocksPerGrid, threadsPerBlock >>>(GPU_A, GPU_B, GPU_C, Size);

	hipMemcpy(C, GPU_C, Size1d * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(GPU_A);
	hipFree(GPU_B);
	hipFree(GPU_C);

#pragma endregion
	for (int i = 0; i < Size1d; i++)
	{
		std::cout << "A: " << A[i] << "	B: " << B[i] << "	c: " << C[i] << std::endl;
	}


	int STOP = 0;
    return 0;
}