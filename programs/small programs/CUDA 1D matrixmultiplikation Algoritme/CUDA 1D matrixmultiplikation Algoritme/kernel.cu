#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <fstream> 

const int MINI_SEC_IN_SEC = 1000;

__global__ void KernelFunc(const int *A, const int *B, int *C, const int Size)
{
    int i = threadIdx.x+ blockIdx.x*blockDim.x;
	int x = i / Size;
	int y = i % Size;

	C[i] = 0;
	//D[i] = (x*Size) + y;
	for (int z = 0; z < Size; z++)
	{
		C[i] += A[(x*Size) + z] * B[(z*Size) + y];
	}
}
int MA(int* A, int* B, int* C, int Size, int Size1d, int max_threadsPerBlock)
{

	int *GPU_A, *GPU_B, *GPU_C;

	hipMalloc((void**)&GPU_A, Size1d * sizeof(int));
	hipMalloc((void**)&GPU_B, Size1d * sizeof(int));
	hipMalloc((void**)&GPU_C, Size1d * sizeof(int));

	hipMemcpy(GPU_A, A, Size1d * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPU_B, B, Size1d * sizeof(int), hipMemcpyHostToDevice);

	// TODO: lab en bedre m�de at g�re dette p�

	int threadsPerBlock = 0;
	int blocksPerGrid = 0;

	if (Size1d< max_threadsPerBlock)
	{
		threadsPerBlock = Size1d;
		blocksPerGrid = 1;
	}
	else
	{
		threadsPerBlock = max_threadsPerBlock;
		blocksPerGrid = (Size1d / max_threadsPerBlock) + 1;
	}

	KernelFunc << <blocksPerGrid, threadsPerBlock >> >(GPU_A, GPU_B, GPU_C, Size);

	hipMemcpy(C, GPU_C, Size1d * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(GPU_A);
	hipFree(GPU_B);
	hipFree(GPU_C);
	return 1;
}

#pragma region Mark
double* Mark3(int* A, int* B, int* C, int max_threadsPerBlock, int Size, int Size1d, int n, int count)
{
	double* result = new double[n];
	double dummy = 0.0;
	for (int j = 0; j<n; j++) {
		clock_t t; // not sure if it is in right format
		t = clock();
		for (int i = 0; i<count; i++)
		{
			dummy += MA(A, B, C, Size, Size1d, max_threadsPerBlock);
		}
		t = clock() - t;
		double time = ((double)t / CLOCKS_PER_SEC)*10;
		result[j] = time;
		std::cout << "time: " << time << " ms" << std::endl;
	}
	return result;
}

double* Mark4(int* A, int* B, int* C, int max_threadsPerBlock, int Size, int Size1d, int n, int count)
{

	double dummy = 0.0;
	double st = 0.0, sst = 0.0;
	for (int j = 0; j<n; j++) {
		clock_t t; // not sure if it is in right format
		t = clock();
		for (int i = 0; i<count; i++)
			dummy += MA(A, B, C, Size, Size1d, max_threadsPerBlock);
		t = clock() - t;
		double time = ((double)t / CLOCKS_PER_SEC)*10;
		st += time;
		sst += time * time;
	}
	double mean = st / n, sdev = sqrt((sst - mean*mean*n) / (n - 1));
	double result[2] = { mean, sdev };
	return result;
}

#pragma endregion


int main()
{
	//const int Size = 3;
	//const int Size1d = Size*Size;
	//int A[Size1d];
	//int B[Size1d];
	//int C[Size1d];
	//int i, x, y, z;
	//hipDeviceProp_t GPU_prop;
	//hipGetDeviceProperties(&GPU_prop, 0);

	//for (int i = 0; i < Size1d; i++)
	//{
	//	A[i] = 2;
	//	B[i] = 3;
	//	C[i] = 0;
	//	std::cout << A[i] << " " << B[i] << std::endl;
	//}
	const int testSize[] { 5, 10, 20, 50, 100, 200, 300, 400, 500, 600, 700, 800, 900, 1000 };
	double result[(sizeof(testSize) / sizeof(*testSize))][3];
	int i, n = 10, count = 100;
	hipDeviceProp_t GPU_prop;
	hipGetDeviceProperties(&GPU_prop, 0);
	int max_threadsPerBlock = GPU_prop.maxThreadsPerBlock;

	for (int i = 0; i < (sizeof(testSize) / sizeof(*testSize)); i++)
	{
		int Size = testSize[i];
		int Size1d = Size * Size;
		int* A = new int[Size1d];
		int* B = new int[Size1d];
		int* C = new int[Size1d];
		for (int x = 0; x < Size1d; x++)
		{
			A[x] = 2;
			B[x] = 3;
		}
		std::cout << testSize[i] << " starting" << std::endl;
		double* Mark4_time = Mark4(A, B, C, max_threadsPerBlock, Size, Size1d, n, count);
		result[i][0] = Size;
		result[i][1] = Mark4_time[0];
		result[i][2] = Mark4_time[1];
	}
#pragma region D1 version
	//int *GPU_A, *GPU_B, *GPU_C;

	//hipMalloc((void**)&GPU_A, Size1d * sizeof(int));
	//hipMalloc((void**)&GPU_B, Size1d * sizeof(int));
	//hipMalloc((void**)&GPU_C, Size1d * sizeof(int));

	//hipMemcpy(GPU_A, A, Size1d * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(GPU_B, B, Size1d * sizeof(int), hipMemcpyHostToDevice);

	//// TODO: lab en bedre m�de at g�re dette p�
	//int threadsPerBlock = Size1d; 
	//int blocksPerGrid = 1;

	//KernelFunc <<<blocksPerGrid, threadsPerBlock >>>(GPU_A, GPU_B, GPU_C, Size);

	//hipMemcpy(C, GPU_C, Size1d * sizeof(int), hipMemcpyDeviceToHost);

	//hipFree(GPU_A);
	//hipFree(GPU_B);
	//hipFree(GPU_C);

#pragma endregion

	//double* result = Mark3(A, B, C, GPU_prop);
	std::ofstream outfile("CUDA_1D_MA_in_C++.txt");
	outfile << "CUDA 1D MA in C++  mean, sdev" << std::endl;

	for (int i = 0; i < (sizeof(testSize) / sizeof(*testSize)); i++)
	{
		outfile << "size: " << result[i][0] << " time: " << result[i][1] << " , " << result[i][2] << std::endl;
	}


	outfile.close();

	int STOP = 0;
    return 0;
}