#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

const int Size = 3;
const int Size1d = Size*Size;
const int n = 10;
const int count = 10000;
const int MINI_SEC_IN_SEC = 1000;

__global__ void KernelFunc(const int *A, const int *B, int *C, const int Size)
{
    int i = threadIdx.x+ blockIdx.x*blockDim.x;
	int x = i / Size;
	int y = i % Size;

	C[i] = 0;
	//D[i] = (x*Size) + y;
	for (int z = 0; z < Size; z++)
	{
		C[i] += A[(x*Size) + z] * B[(z*Size) + y];
	}
}
int MA(int* A, int* B, int* C, int Size, int Size1d, hipDeviceProp_t GPU_prop)
{

	int *GPU_A, *GPU_B, *GPU_C;

	hipMalloc((void**)&GPU_A, Size1d * sizeof(int));
	hipMalloc((void**)&GPU_B, Size1d * sizeof(int));
	hipMalloc((void**)&GPU_C, Size1d * sizeof(int));

	hipMemcpy(GPU_A, A, Size1d * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(GPU_B, B, Size1d * sizeof(int), hipMemcpyHostToDevice);

	// TODO: lab en bedre m�de at g�re dette p�

	int threadsPerBlock = 0;
	int blocksPerGrid = 0;

	if (Size1d< GPU_prop.maxThreadsPerBlock)
	{
		threadsPerBlock = Size1d;
		blocksPerGrid = 1;
	}
	else
	{
		threadsPerBlock = GPU_prop.maxThreadsPerBlock;
		blocksPerGrid = (Size1d/GPU_prop.maxThreadsPerBlock)+1;
	}

	KernelFunc << <blocksPerGrid, threadsPerBlock >> >(GPU_A, GPU_B, GPU_C, Size);

	hipMemcpy(C, GPU_C, Size1d * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(GPU_A);
	hipFree(GPU_B);
	hipFree(GPU_C);
	return 1;
}

#pragma region Mark
double* Mark3(int* A, int* B, int* C, hipDeviceProp_t GPU_prop)
{
	double result[n];
	double dummy = 0.0;
	for (int j = 0; j<n; j++) {
		clock_t t; // not sure if it is in right format
		t = clock();
		for (int i = 0; i<count; i++)
		{
			dummy += MA(A, B, C, Size, Size1d, GPU_prop);
		}
		t = clock() - t;
		double time = ((double)t / CLOCKS_PER_SEC)*MINI_SEC_IN_SEC;
		result[j] = time;
		std::cout << "time: " << time << " ms" << std::endl;
	}
	return result;
}
#pragma endregion


int main()
{
	const int Size = 3;
	const int Size1d = Size*Size;
	int A[Size1d];
	int B[Size1d];
	int C[Size1d];
	int i, x, y, z;

	for (int i = 0; i < Size1d; i++)
	{
		A[i] = 2;
		B[i] = 3;
		C[i] = 0;
		std::cout << A[i] << " " << B[i] << std::endl;
	}
#pragma region D1 version
	//int *GPU_A, *GPU_B, *GPU_C;

	//hipMalloc((void**)&GPU_A, Size1d * sizeof(int));
	//hipMalloc((void**)&GPU_B, Size1d * sizeof(int));
	//hipMalloc((void**)&GPU_C, Size1d * sizeof(int));

	//hipMemcpy(GPU_A, A, Size1d * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(GPU_B, B, Size1d * sizeof(int), hipMemcpyHostToDevice);

	//// TODO: lab en bedre m�de at g�re dette p�
	//int threadsPerBlock = Size1d; 
	//int blocksPerGrid = 1;

	//KernelFunc <<<blocksPerGrid, threadsPerBlock >>>(GPU_A, GPU_B, GPU_C, Size);

	//hipMemcpy(C, GPU_C, Size1d * sizeof(int), hipMemcpyDeviceToHost);

	//hipFree(GPU_A);
	//hipFree(GPU_B);
	//hipFree(GPU_C);

#pragma endregion
	hipDeviceProp_t GPU_prop;
	hipGetDeviceProperties(&GPU_prop, 0);
	double* result = Mark3(A, B, C, GPU_prop);

	for (int i = 0; i < Size1d; i++)
	{
		std::cout << "A: " << A[i] << "	B: " << B[i] << "	c: " << C[i] << std::endl;
	}


	int STOP = 0;
    return 0;
}